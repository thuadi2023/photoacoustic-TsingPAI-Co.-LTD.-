#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"  
#include ""    
#include "stdafx.h"
#include "cdefaultbuild.cuh"


__global__ void recKernel(GPUMemory gpu_memory, RecoParam reco_param)
{

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int k = j * 512 + i;

	float dx = 0;
	float dy = 0;
	float rr0 = 0;
	float idx_temp = 0;
	float pa_temp = 0;
	float new_a = 0;

	float Vs = reco_param.voice_speed;
	float Fr = reco_param.sampe_fr;
	int Tp = reco_param.time_point;
	float R = reco_param.radius;


	float pa = 0;
	float p = 0;
	float pa1 = 0;
	float pa2 = 0;
	float pa3 = 0;
	float pa4 = 0;
	
	int int_idx = 0;
	
      float pi = 3.14159265359;
	
	float paDiff;
	float paBP;
	float angleCorrection;
	float dOmega;


	for (int iStep = 0; iStep < reco_param.channel; iStep++)
	{
		dx = (k % 512 + 1 - 512 / 2.0)*reco_param.pixel_size - (iStep - reco_param.channel / 2) * reco_param.pitch;
		dy = ((512 - k / 512) - 512 / 2.0)*reco_param.pixel_size - 512 / 2 * reco_param.pixel_size;
		if (fabs(dx) < fabs(dy))
		{
			rr0 = sqrt(dx * dx + dy * dy);
			idx_temp = rr0 / Vs * Fr+74;
			if (idx_temp > (Tp - 1 - 4))
			{
				idx_temp = Tp - 1 - 4;
			}
			else if (idx_temp < 1)
			{
				idx_temp = 1;
			}

			int_idx = int(idx_temp);
			pa1 = gpu_memory.Rawdata[(int_idx)+iStep * Tp  - 1];
			pa2 = gpu_memory.Rawdata[(int_idx)+iStep * Tp ];
			pa23 = gpu_memory.Rawdata[(int_idx)+iStep * Tp + 1 ];
			paDiff = pa2 - pa1;
			paBP = 2 * pa1;
			rr0 = rr0 + 0.001;
			dOmega = 0.25*1e-3*0.25*1e-3* fabs(dy) / (rr0 * rr0 * rr0);
			angleCorrection = dOmega / (2 * pi);
			pa = pa + pa2;
		}
		else
		{
			pa = pa + 0;
		}
	}



	new_a = (((exp(((reco_param.scanDepth[k / 512 / 96 + 2] - reco_param.scanDepth[k / 512 / 96 + 1])/96.0*(k / 512 % 96)+reco_param.scanDepth[k / 512 / 96 + 1])*0.1) + std::exp(j / 150.0)))* exp((reco_param.amp - 50)*0.2))*0.1;
	pa_temp = pa * new_a / 200 + (reco_param.dynamicRange - 50) * 5;
	if (pa_temp > 255)
		pa_temp = 255;
	else if (pa_temp < 0)
		pa_temp = 0;
	gpu_memory.ImageArray[k] = uchar(pa_temp);
}

